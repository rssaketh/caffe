#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

// ----------------------------------------------------------------------------
// ------------------- Begin caffe wrappers for cusolverDn --------------------
// ----------------------------------------------------------------------------

template <>
void caffe_gpu_inverse_qr<float>(const CBLAS_SIDE SideA,
    const CBLAS_TRANSPOSE TransA, const int M, const int N, const float alpha,
    float* const A, float* const TAU, float* const B, const int Lwork,
    float* const Workspace, int* const devInfo) {
  // Note: cusolverDn uses fortran-order
  const int ldb = N;
  int lda = (SideA == CblasLeft) ? M : N;
  hipblasSideMode_t cuSideA =
      (SideA == CblasLeft) ? HIPBLAS_SIDE_RIGHT : HIPBLAS_SIDE_LEFT;
  hipblasOperation_t cuTransR =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransQ =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  // First generate QR factorization of A
  CUSOLVER_CHECK(hipsolverDnSgeqrf(Caffe::cusolver_dn_handle(), lda, lda, A,
      lda, TAU, Workspace, Lwork, devInfo));
  if ((SideA == CblasLeft && TransA == CblasNoTrans) ||
      (SideA == CblasRight && TransA == CblasTrans)) {
    // First do trsm and then ormqr
    // Pre/Post-Multiply B by inverse of R or R'
    CUBLAS_CHECK(hipblasStrsm(Caffe::cublas_handle(), cuSideA,
        HIPBLAS_FILL_MODE_UPPER, cuTransR, HIPBLAS_DIAG_NON_UNIT, N, M, &alpha,
        A, lda, B, ldb));
    // Post/Pre-Multiply B by inverse of Q' or Q without generating Q
    CUSOLVER_CHECK(hipsolverDnSormqr(Caffe::cusolver_dn_handle(), cuSideA,
      cuTransQ, N, M, lda, A, lda, TAU, B, ldb, Workspace, Lwork, devInfo));
  } else {
    // First do ormqr and then trsm
    // Pre/Post-Multiply B by inverse of Q or Q' without generating Q
    CUSOLVER_CHECK(hipsolverDnSormqr(Caffe::cusolver_dn_handle(), cuSideA,
      cuTransQ, N, M, lda, A, lda, TAU, B, ldb, Workspace, Lwork, devInfo));
    // Post/Pre-Multiply B by inverse of R' or R
    CUBLAS_CHECK(hipblasStrsm(Caffe::cublas_handle(), cuSideA,
        HIPBLAS_FILL_MODE_UPPER, cuTransR, HIPBLAS_DIAG_NON_UNIT, N, M, &alpha,
        A, lda, B, ldb));
  }
}

template <>
void caffe_gpu_inverse_qr<double>(const CBLAS_SIDE SideA,
    const CBLAS_TRANSPOSE TransA, const int M, const int N, const double alpha,
    double* const A, double* const TAU, double* const B, const int Lwork,
    double* const Workspace, int* const devInfo) {
  // Note: cusolverDn uses fortran-order
  const int ldb = N;
  int lda = (SideA == CblasLeft) ? M : N;
  hipblasSideMode_t cuSideA =
      (SideA == CblasLeft) ? HIPBLAS_SIDE_RIGHT : HIPBLAS_SIDE_LEFT;
  hipblasOperation_t cuTransR =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransQ =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  // First generate QR factorization of A
  CUSOLVER_CHECK(hipsolverDnDgeqrf(Caffe::cusolver_dn_handle(), lda, lda, A,
      lda, TAU, Workspace, Lwork, devInfo));
  if ((SideA == CblasLeft && TransA == CblasNoTrans) ||
      (SideA == CblasRight && TransA == CblasTrans)) {
    // First do trsm and then ormqr
    // Pre/Post-Multiply B by inverse of R or R'
    CUBLAS_CHECK(hipblasDtrsm(Caffe::cublas_handle(), cuSideA,
        HIPBLAS_FILL_MODE_UPPER, cuTransR, HIPBLAS_DIAG_NON_UNIT, N, M, &alpha,
        A, lda, B, ldb));
    // Post/Pre-Multiply B by inverse of Q' or Q without generating Q
    CUSOLVER_CHECK(hipsolverDnDormqr(Caffe::cusolver_dn_handle(), cuSideA,
      cuTransQ, N, M, lda, A, lda, TAU, B, ldb, Workspace, Lwork, devInfo));
  } else {
    // First do ormqr and then trsm
    // Pre/Post-Multiply B by inverse of Q or Q' without generating Q
    CUSOLVER_CHECK(hipsolverDnDormqr(Caffe::cusolver_dn_handle(), cuSideA,
      cuTransQ, N, M, lda, A, lda, TAU, B, ldb, Workspace, Lwork, devInfo));
    // Post/Pre-Multiply B by inverse of R' or R
    CUBLAS_CHECK(hipblasDtrsm(Caffe::cublas_handle(), cuSideA,
        HIPBLAS_FILL_MODE_UPPER, cuTransR, HIPBLAS_DIAG_NON_UNIT, N, M, &alpha,
        A, lda, B, ldb));
  }
}

template <>
void caffe_gpu_orthogonalize<float>(const int M, const int N,
    float* const A, float* const TAU, const int Lwork,
    float* const Workspace, int* const devInfo) {
  // A is M * N (i.e. M rows, N columns) in row major with N >= M.
  const int lda = N;
  // First generate QR factorization of transpose(A)
  CUSOLVER_CHECK(hipsolverDnSgeqrf(Caffe::cusolver_dn_handle(), N, M, A,
      lda, TAU, Workspace, Lwork, devInfo));
  // Generate orthogonal Q matrix and overwrite its transpose on A.
  CUSOLVER_CHECK(hipsolverDnSorgqr(Caffe::cusolver_dn_handle(), N, M, M, A,
      lda, TAU, Workspace, Lwork, devInfo));
}

template <>
void caffe_gpu_orthogonalize<double>(const int M, const int N,
    double* const A, double* const TAU, const int Lwork,
    double* const Workspace, int* const devInfo) {
  // A is M * N (i.e. M rows, N columns) in row major with N >= M.
  const int lda = N;
  // First generate QR factorization of transpose(A)
  CUSOLVER_CHECK(hipsolverDnDgeqrf(Caffe::cusolver_dn_handle(), N, M, A,
      lda, TAU, Workspace, Lwork, devInfo));
  // Generate orthogonal Q matrix and overwrite its transpose on A.
  CUSOLVER_CHECK(hipsolverDnDorgqr(Caffe::cusolver_dn_handle(), N, M, M, A,
      lda, TAU, Workspace, Lwork, devInfo));
}

template <>
void caffe_gpu_buffersize_qr<float>(const int M, const int N,
    float* const A, float* const TAU, int* const Lwork) {
  const int lda = M;
  int lwork_geqrf = 0, lwork_orgqr = 0;
  CUSOLVER_CHECK(hipsolverDnSgeqrf_bufferSize(Caffe::cusolver_dn_handle(),
      M, N, A, lda, &lwork_geqrf));
  CUSOLVER_CHECK(hipsolverDnSorgqr_bufferSize(Caffe::cusolver_dn_handle(),
      M, N, N, A, lda, TAU, &lwork_orgqr));
  *Lwork = (lwork_geqrf > lwork_orgqr) ? lwork_geqrf : lwork_orgqr;
}

template <>
void caffe_gpu_buffersize_qr<double>(const int M, const int N,
    double* const A, double* const TAU, int* const Lwork) {
  const int lda = M;
  int lwork_geqrf = 0, lwork_orgqr = 0;
  CUSOLVER_CHECK(hipsolverDnDgeqrf_bufferSize(Caffe::cusolver_dn_handle(),
      M, N, A, lda, &lwork_geqrf));
  CUSOLVER_CHECK(hipsolverDnDorgqr_bufferSize(Caffe::cusolver_dn_handle(),
      M, N, N, A, lda, TAU, &lwork_orgqr));
  *Lwork = (lwork_geqrf > lwork_orgqr) ? lwork_geqrf : lwork_orgqr;
}

// ----------------------------------------------------------------------------
// -------------------- End caffe wrappers for cusolverDn ---------------------
// ----------------------------------------------------------------------------

template <typename Dtype>
__global__ void absymm_kernel(const int n, const Dtype alpha, const Dtype beta,
    const Dtype* const x, Dtype* const y) {
  CUDA_KERNEL_LOOP(index, n * n) {
    y[index] = (alpha * x[index]) + (beta * x[((index % n)* n) + (index / n)]);
  }
}

template <>
void caffe_gpu_absymm<float>(const int N, const float alpha, const float beta,
    const float* const A, float* const B) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  absymm_kernel<float><<<CAFFE_GET_BLOCKS(N * N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, beta, A, B);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_absymm<double>(const int N, const double alpha,
    const double beta, const double* const A, double* const B) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  absymm_kernel<double><<<CAFFE_GET_BLOCKS(N * N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, beta, A, B);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float* X,
                           hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double* X,
                            hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}

template <>
void caffe_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_asum<float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
void caffe_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template void caffe_gpu_set<int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double>(const int N, const double alpha, double* Y);

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template <>
void caffe_gpu_add<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template <>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

template <>
void caffe_gpu_abs<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}


template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template <>
void caffe_gpu_exp<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template <>
void caffe_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <typename Dtype>
__global__ void sqrt_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = sqrt(a[index]);
  }
}

template <>
void caffe_gpu_sqrt<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_sqrt<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
                                      - (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}

}  // namespace caffe
