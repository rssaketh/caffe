#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/custom_conv_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void replace_elements(const int nhwc, Dtype* in, int middle, int value) {
  int j =  blockIdx.x*blockDim.x + middle;
  if (j < nhwc)
    in[j] = value;
}

template <typename Dtype>
__global__ void normalize_kernel(const int nhwc, Dtype* in, int num_threads) { 
  if (threadIdx.x == 0) { 
    int sum=0;
    for (int i=0;i<num_threads;++i)
      sum+= in[blockIdx.x*blockDim.x + i];

    for (int j=0;j < num_threads;++j)
      in[j] = in[j]/sum;
  
  }
}
  



template <typename Dtype>
void CustomConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

//Code added starts here
  //Dtype* weight_dum = this->blobs_[0]->mutable_gpu_data();
  int n = this->blobs_[0]->shape(0),k=this->blobs_[0]->shape(1),
      h = this->blobs_[0]->shape(2),w = this->blobs_[0]->shape(3);
  
  replace_elements<Dtype><<<n*k, w*h>>>(n*k*w*h,this->blobs_[0]->mutable_gpu_data(),(w*h)/2,0);
  normalize_kernel<Dtype><<<n*k, w*h>>>(n*k*w*h,this->blobs_[0]->mutable_gpu_data(),w*h);
  replace_elements<Dtype><<<n*k, w*h>>>(n*k*w*h,this->blobs_[0]->mutable_gpu_data(),(w*h)/2,-1);



//Code added ends here
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
          top_data + n * this->top_dim_);
      if (this->bias_term_) {
        const Dtype* bias = this->blobs_[1]->gpu_data();
        this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
      }
    }
  }
}

template <typename Dtype>
void CustomConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CustomConvolutionLayer);

}  // namespace caffe
